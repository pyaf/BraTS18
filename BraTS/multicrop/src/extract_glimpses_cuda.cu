#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ATen/ATen.h"

// for cuda::type<scalar_t>;
#include "ATen/cuda/CUDATypeConversion.cuh"
// line 107, 303, 316 for lambda syntax

#include "utils.h"

template <typename T>
__global__ void Crop2DFKernel(
    const int numels,
    const T* image,
    const int16_t * fixs,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int stride,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(i, numels) {
    int w = i % pooled_width;
    int h = (i / pooled_width) % pooled_height;
    int c = (i / pooled_width / pooled_height) % channels;
    int n =  i / pooled_width / pooled_height / channels;

    const int16_t* pos = fixs + 2*n;
    int row = pos[0] - (pooled_height/2 - h)*stride;
    int col = pos[1] - (pooled_width/2  - w)*stride;

    if (row < 0) row = 0;
    if (row >= height) row = height - 1;

    if (col < 0) col = 0;
    if (col >= width) col = width - 1;

    int j = c * height * width + row * width + col;


    top_data[i] = image[j];
  }
}


template <typename T>
__global__ void Crop2DLKernel(
    const int numels,
    const T* image,
    const int16_t * fixs,
    const int height,
    const int width,
    const int channels,
    const int pooled_height,
    const int pooled_width,
    const int stride,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(i, numels) {
    int c = i % channels;
    int w = (i / channels) % pooled_width;
    int h = (i / channels / pooled_width ) % pooled_height;
    int n =  i / channels / pooled_width / pooled_height;

    const int16_t* pos = fixs + 2*n;
    int row = pos[0] - (pooled_height/2 - h)*stride;
    int col = pos[1] - (pooled_width/2  - w)*stride;

    if (row < 0) row = 0;
    if (row >= height) row = height - 1;

    if (col < 0) col = 0;
    if (col >= width) col = width - 1;

    int j = row * width* channels + col * channels + c;

    top_data[i] = image[j];
  }
}

at::Tensor crop2d_gpu(
    const at::Tensor &X, // 3d image hwc
    const at::Tensor &R, // boxes
    int pooled_height, int pooled_width,
    int stride=1, bool first=false
    ) {

  at::Tensor output;
  int channels, off=0;

  if (X.dim() == 2) {
    channels = 1;
    off = 1;
    output = X.type().zeros(
        {R.size(0), pooled_height, pooled_width});
  } else if (first) {
    channels = X.size(0);
    output = X.type().zeros(
        {R.size(0), channels, pooled_height, pooled_width});
  } else {
    channels = X.size(2);
    output = X.type().zeros(
        {R.size(0), pooled_height, pooled_width, channels});
  }

  const int output_size = output.numel();
  const int threads = 1024;
  const int blocks = (output_size + threads - 1) / threads;

  if (first) {
    AT_DISPATCH_ALL_TYPES(X.type(), "crop2d_cuda", [&] {
        using cuda_scalar_t = at::cuda::type<scalar_t>;
        Crop2DFKernel<cuda_scalar_t>
          <<<blocks, threads>>>(
              output_size,
              X.data<cuda_scalar_t>(),
              R.data<int16_t>(),
              X.size(1-off),
              X.size(2-off),
              channels,
              pooled_height,
              pooled_width,
              stride,
              output.data<cuda_scalar_t>());
          });
  } else {
    AT_DISPATCH_ALL_TYPES(X.type(), "crop2d_cuda", [&] {
        using cuda_scalar_t = at::cuda::type<scalar_t>;
        Crop2DLKernel<cuda_scalar_t>
          <<<blocks, threads>>>(
              output_size,
              X.data<cuda_scalar_t>(),
              R.data<int16_t>(),
              X.size(0),
              X.size(1),
              channels,
              pooled_height,
              pooled_width,
              stride,
              output.data<cuda_scalar_t>());
          });
  }
  return output;
}


template <typename T>
__global__ void Crop3DFKernel(
    const int numels,
    const T* image,
    const int16_t * fixs,
    const int length,
    const int height,
    const int width,
    const int channels,
    const int pooled_length,
    const int pooled_height,
    const int pooled_width,
    const int stride,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(i, numels) {
    int w = i % pooled_width;
    int h = (i / pooled_width) % pooled_height;
    int l = (i / pooled_width / pooled_height) % pooled_length;
    int c = (i / pooled_width / pooled_height / pooled_length) % channels;
    int n = i / pooled_width / pooled_height / pooled_length / channels;

    const int16_t * pos = fixs + 3*n;
    int len = pos[0] - (pooled_length/2 - l)*stride;
    int row = pos[1] - (pooled_height/2 - h)*stride;
    int col = pos[2] - (pooled_width/2  - w)*stride;

    if (len < 0) len = 0;
    if (len >= length) len = length - 1;

    if (row < 0) row = 0;
    if (row >= height) row = height - 1;

    if (col < 0) col = 0;
    if (col >= width) col = width - 1;


    int j = c * length * height * width + len * height * width + row * width + col;

    top_data[i] = image[j];
  }
}

template <typename T>
__global__ void Crop3DLKernel(
    const int numels,
    const T* image,
    const int16_t * fixs,
    const int length,
    const int height,
    const int width,
    const int channels,
    const int pooled_length,
    const int pooled_height,
    const int pooled_width,
    const int stride,
    T* top_data) {
  CUDA_1D_KERNEL_LOOP(i, numels) {
    int c = i % channels;
    int w = (i / channels) % pooled_width;
    int h = (i / channels / pooled_width ) % pooled_height;
    int l = (i / channels / pooled_width / pooled_height) % pooled_length;
    int n =  i / channels / pooled_width / pooled_height / pooled_length;

    const int16_t * pos = fixs + 3*n;
    int len = pos[0] - (pooled_length/2 - l)*stride;
    int row = pos[1] - (pooled_height/2 - h)*stride;
    int col = pos[2] - (pooled_width/2  - w)*stride;

    if (len < 0) len = 0;
    if (len >= length) len = length - 1;

    if (row < 0) row = 0;
    if (row >= height) row = height - 1;

    if (col < 0) col = 0;
    if (col >= width) col = width - 1;


    int j = len*height*width*channels + row * width* channels + col * channels + c;

    top_data[i] = image[j];
  }
}


at::Tensor crop3d_gpu(
    const at::Tensor &X, // 4d image thwc
    const at::Tensor &R, // boxes
    int pooled_length, int pooled_height, int pooled_width,
    int stride=1, bool first=false
    ) {

  at::Tensor output;
  int channels, off=0;
  if (X.dim() == 3) {
    channels = 1;
    off = 1;
    output = X.type().zeros(
        {R.size(0), pooled_length, pooled_height, pooled_width});
  } else if (first) {
    channels = X.size(0);
    output = X.type().zeros(
        {R.size(0), channels, pooled_length, pooled_height, pooled_width});
  } else {
    channels = X.size(3);
    output = X.type().zeros(
        {R.size(0), pooled_length, pooled_height, pooled_width, channels});
  }

  const int output_size = output.numel();
  const int threads = 1024;
  const int blocks = (output_size + threads - 1) / threads;

  if (first) {
    AT_DISPATCH_ALL_TYPES(X.type(), "crop3d_cuda", [&] {
        using cuda_scalar_t = at::cuda::type<scalar_t>;
        Crop3DFKernel<cuda_scalar_t>
          <<<blocks, threads>>>(
              output_size,
              X.data<cuda_scalar_t>(),
              R.data<int16_t>(),
              X.size(1-off),
              X.size(2-off),
              X.size(3-off),
              channels,
              pooled_length,
              pooled_height,
              pooled_width,
              stride,
              output.data<cuda_scalar_t>());
          });
  } else {
    AT_DISPATCH_ALL_TYPES(X.type(), "crop3d_cuda", [&] {
        using cuda_scalar_t = at::cuda::type<scalar_t>;
        Crop3DLKernel<cuda_scalar_t>
          <<<blocks, threads>>>(
              output_size,
              X.data<cuda_scalar_t>(),
              R.data<int16_t>(),
              X.size(0),
              X.size(1),
              X.size(2),
              channels,
              pooled_length,
              pooled_height,
              pooled_width,
              stride,
              output.data<cuda_scalar_t>());
          });
  }

  return output;
}

//void crop2d_gpu(
//    const at::Tensor &X, // 3d image hwc
//    const at::Tensor &R, // boxes
//    int pooled_height, int pooled_width,
//    int stride=1, bool first=false
//    ) {
//  //cuda::type<float> x;
//  AT_DISPATCH_ALL_TYPES(
//      X.type(), "crop2d", [&](){
//      using cuda_scalar_t = at::cuda::type<scalar_t>;
//      //scalar_t s = 1;
//      });
//}
//
//at::Tensor crop2d_gpu(
//    const at::Tensor &X, // 3d image hwc
//    const at::Tensor &R, // boxes
//    int pooled_height, int pooled_width,
//    int stride=1, bool first=false
//    ) {
//  return AT_DISPATCH_ALL_TYPES(
//      X.type(), "crop2d", [&]() -> at::Tensor {
//      return _crop2d_gpu<at::cuda::type<scalar_t>>(
//          X, R, pooled_height, pooled_width, stride, first);
//      });
//}
//
//at::Tensor crop3d_gpu(
//    const at::Tensor &X, // 4d image thwc
//    const at::Tensor &R, // boxes
//    int pooled_length, int pooled_height, int pooled_width,
//    int stride=1, bool first=false
//    ) {
//  return AT_DISPATCH_ALL_TYPES(X.type(), "crop3d", [&]() -> at::Tensor {
//      return _crop3d_gpu<at::cuda::type<scalar_t>>(
//          X, R, pooled_length, pooled_height, pooled_width, stride, first);
//      });
//}
